#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <errno.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#include <malloc.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKWIDTH 1024

int *allocateMemoryInt(int length) {

   int *vec;

   if ((vec = (int *)malloc(length * sizeof(int))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }
   memset(vec, 0, length * sizeof(int));
   return vec;
}
double *allocateMemoryDouble(int length) {

   double *vec;

   if ((vec = (double *)malloc(length * sizeof(double))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }
   memset(vec, 0, length * sizeof(double));
   return vec;
}

__global__ void CalcPageRank(int nodes, int edges, int *in_d, int *out_d, 
      int *run_d, int *edges_d, double *pagerank_old_d, double *pagerank_new_d)
{
   int node_index = blockIdx.x * BLOCKWIDTH + threadIdx.x;
   double sum = 0;
   double d = 0.85;
   double jumpChance = (1 - d) * (1.0 / nodes);
   int stopIdx = run_d[node_index] + in_d[node_index];
   int k;
   
   for (k = run_d[node_index]; k < stopIdx; k++) {
      int jk = edges_d[k];
      sum += pagerank_old_d[jk] / out_d[jk];
   }
   
   pagerank_new_d[node_index] = sum * d + jumpChance;
   __syncthreads();
   pagerank_old_d[node_index] = pagerank_new_d[node_index];
}

int main (int argc, char **argv) {

   if(argc!=4) {
      printf("Invalid Syntax <Nodecount> <Edgecount> <Itercount>\n");
      return 0;
   }
   int i = 0, j = 0, k = 0, run = 0, idx = 0;
   int nodes=atoi(argv[1]);
   int edges=atoi(argv[2]);
   int iter = atoi(argv[3]);
   struct timeval stop, start;

   int*  indegree_count=allocateMemoryInt(nodes);
   int*  outdegree_count=allocateMemoryInt(nodes);
   int*  running_edge_indices=allocateMemoryInt(nodes);
   int*  edges_1D = allocateMemoryInt(edges);//node1:node2|node2->node1

   double*  pagerank_new=allocateMemoryDouble(nodes);
   double*  pagerank_old=allocateMemoryDouble(nodes);   

   for (i = 0; i < nodes; i++)
      pagerank_old[i] = 1 / (double)nodes;
   
   gettimeofday(&start, NULL);

   setvbuf(stdin, NULL, _IOFBF, edges);

   //reads in edges
   for (i = 0; i < edges; i++) {
      scanf("%d\n", &j);
      edges_1D[i] = j;
   }

   gettimeofday(&stop, NULL);
   fprintf(stderr, "took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));

   //reads in in-degrees, out-degrees, and computes running idx
   for (i = 0; i < nodes; i++) {
      scanf("%d %d %d\n", &idx, &j, &k);
      indegree_count[idx] = j;
      outdegree_count[idx] = k;
      running_edge_indices[idx] = run;
      run += j;
   }

   gettimeofday(&stop, NULL);
   fprintf(stderr, "took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));

   int *in_d, *out_d, *run_d, *edges_d;
   double *pagerank_new_d, *pagerank_old_d;
   
   int node_size = nodes * sizeof(int);
   int pr_size = nodes * sizeof(double);
   int edges_size = edges * sizeof(int);
   
   hipMalloc(&in_d, node_size);
   hipMemcpy(in_d, indegree_count, node_size, hipMemcpyHostToDevice);
   
   hipMalloc(&out_d, node_size);
   hipMemcpy(out_d, outdegree_count, node_size, hipMemcpyHostToDevice);
   
   hipMalloc(&run_d, node_size);
   hipMemcpy(run_d, running_edge_indices, node_size, hipMemcpyHostToDevice);
   
   hipMalloc(&edges_d, edges_size);
   hipMemcpy(edges_d, edges_1D, edges_size, hipMemcpyHostToDevice);
   
   hipMalloc(&pagerank_old_d,pr_size);
   hipMemcpy(pagerank_old_d, pagerank_old, pr_size, hipMemcpyHostToDevice);
   
   hipMalloc(&pagerank_new_d,pr_size);
   hipMemcpy(pagerank_new_d, pagerank_new, pr_size, hipMemcpyHostToDevice);

   int blocks = ceil((double)nodes/(double)BLOCKWIDTH);
   dim3 dimGrid(blocks, 1);
   dim3 dimBlock(BLOCKWIDTH, 1);
   for(i=0; i < iter; i++)
   {
      CalcPageRank<<<dimGrid, dimBlock>>>(nodes, edges, in_d, out_d, run_d, edges_d, 
         pagerank_old_d, pagerank_new_d);
   }

   hipMemcpy(pagerank_old, pagerank_old_d, pr_size, hipMemcpyDeviceToHost);
   gettimeofday(&stop, NULL);
   fprintf(stderr, "took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));
      
   for (i = 0; i < nodes; i++)
      printf("%.15lf:%d,", pagerank_old[i], i);

   hipFree(in_d);
   hipFree(out_d);
   hipFree(run_d);
   hipFree(edges_d);
   hipFree(pagerank_old_d);
   hipFree(pagerank_new_d);

   return 0;
}






