
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <errno.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#include <malloc.h>
#include <math.h>

#define BLOCKWIDTH 1024
int *allocateMemoryInt(int length) {

   int *vec;

   if ((vec = (int *)malloc(length * sizeof(int))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }
   memset(vec, 0, length * sizeof(int));
   return vec;
}
double *allocateMemoryDouble(int length) {

   double *vec;

   if ((vec = (double *)malloc(length * sizeof(double))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }
   memset(vec, 0, length * sizeof(double));
   return vec;
}

__global__ void CalcPageRank(int nodes, int edges, int *in_d, int *out_d, 
      int *run_d, int *edges_d, double *pagerank_old_d, double *pagerank_new_d)
{
   int node_index = blockIdx.x * BLOCKWIDTH + threadIdx.x;
   double sum = 0;
   double d = 0.85;
   double jumpChance = (1 - d) * (1.0 / nodes);
   int stopIdx = run_d[node_index] + in_d[node_index];
   int k;
   for (k = run_d[node_index]; k < stopIdx; k++) {
      int jk = edges_d[k];
      sum += pagerank_old_d[jk] / out_d[jk];
   }
   pagerank_new_d[node_index] = sum * d + jumpChance;
   __syncthreads();
   pagerank_old_d[node_index]=pagerank_new_d[node_index];
}

int main (int argc, char **argv) {

   if(argc!=4) {
      printf("Invalid Syntax <Nodecount> <Edgecount> <Itercount>\n");
      return 0;
   }
   int i = 0, j = 0, k = 0, run = 0, idx = 0;
   int nodes=atoi(argv[1]);
   int edges=atoi(argv[2]);
   int iter = atoi(argv[3]);
   struct timeval stop, start;

   

   int*  indegree_count=allocateMemoryInt(nodes);
   int*  outdegree_count=allocateMemoryInt(nodes);
   int*  running_edge_indices=allocateMemoryInt(nodes + 1);
   int*  edges_1D = allocateMemoryInt(edges);//node1:node2|node2->node1

   double*  pagerank_new=allocateMemoryDouble(nodes);
   double*  pagerank_old=allocateMemoryDouble(nodes);   
   

   for (i = 0; i < nodes; i++)
      pagerank_old[i] = 1 / (double)nodes;
   
   gettimeofday(&start, NULL);

   fprintf(stderr, "%d\n", setvbuf(stdin, NULL, _IOFBF, edges));

   //reads in edges
   for (i = 0; i < edges; i++) {
      scanf("%d\n", &j);
      edges_1D[i] = j;
   }

   gettimeofday(&stop, NULL);
   fprintf(stderr, "took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));

   //reads in in-degrees, out-degrees, and computes running idx
   for (i = 0; i < nodes; i++) {
      scanf("%d %d %d\n", &idx, &j, &k);
      indegree_count[idx] = j;
      outdegree_count[idx] = k;
      running_edge_indices[idx] = run;
      run += j;
   }

   gettimeofday(&stop, NULL);
   fprintf(stderr, "took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));



   int *in_d, *out_d, *run_d, *edges_d;
   double *pagerank_new_d, *pagerank_old_d;
   hipMalloc(&in_d, nodes);
   hipMemcpy(in_d, indegree_count, nodes, hipMemcpyHostToDevice);
   hipMalloc(&out_d, nodes);
   hipMemcpy(out_d, outdegree_count, nodes, hipMemcpyHostToDevice);
   hipMalloc(&run_d, nodes+1);
   hipMemcpy(run_d, running_edge_indices, nodes+1, hipMemcpyHostToDevice);
   hipMalloc(&edges_d, edges);
   hipMemcpy(edges_d, edges_1D, edges, hipMemcpyHostToDevice);
   hipMalloc(&pagerank_old_d,nodes);
   hipMemcpy(pagerank_old_d, pagerank_old, nodes, hipMemcpyHostToDevice);
   hipMalloc(&pagerank_new_d,nodes);
   hipMemcpy(pagerank_new_d, pagerank_new, nodes, hipMemcpyHostToDevice);

   int blocks = ceil((double)nodes/(double)BLOCKWIDTH);
   dim3 dimGrid(blocks, 1);
   dim3 dimBlock(BLOCKWIDTH, 1);
   for(i=0; i < nodes; i++)
   {
      CalcPageRank<<<dimGrid, dimBlock>>>(nodes, edges, in_d, out_d, run_d, edges_d, 
         pagerank_old_d, pagerank_new_d);
   }

   hipMemcpy(pagerank_old, pagerank_old_d, nodes, hipMemcpyDeviceToHost);
   gettimeofday(&stop, NULL);
   fprintf(stderr, "took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));
    for (i = 0; i < nodes; i++)
      printf("%.15lf:%d,", pagerank_old[i], i);

   hipFree(in_d);
   hipFree(out_d);
   hipFree(run_d);
   hipFree(edges_d);
   hipFree(pagerank_old_d);
   hipFree(pagerank_new_d);

   for (i = 0; i < nodes; i++)
      printf("%.15lf:%d,", pagerank_old[i], i);

   free(indegree_count);
   free(outdegree_count);
   free(pagerank_new);
   free(pagerank_old);
   free(running_edge_indices);
   free(edges_1D);
   return 0;

}






