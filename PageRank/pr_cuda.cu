#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <errno.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#include <malloc.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKWIDTH 1024

int *allocateMemoryInt(int length) {

   int *vec;

   if ((vec = (int *)malloc(length * sizeof(int))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }
   memset(vec, 0, length * sizeof(int));
   return vec;
}


// CUDA Kernal Call
__global__ void CalcPageRank(int nodes, int edges, int *in_d, int *out_d, 
      int *run_d, int *edges_d, double *pagerank_old_d, double *pagerank_new_d) {
   
   int node_index = blockIdx.x * BLOCKWIDTH + threadIdx.x; 

   if(node_index < nodes) {

      double sum = 0;
      double d = 0.85;
      double jumpChance = (1 - d) * (1.0 / nodes);
      int k;

      //move along edges array for each node pointing at the current node
      int stopIdx = run_d[node_index] + in_d[node_index];
      
      for (k = run_d[node_index]; k < stopIdx; k++) {
         int jk = edges_d[k];   // get a node pointing at the current node
         sum += pagerank_old_d[jk] / out_d[jk];
      }  
   
      pagerank_new_d[node_index] = sum * d + jumpChance;
   }
   __syncthreads();
   pagerank_old_d[node_index] = pagerank_new_d[node_index];
}

int main (int argc, char **argv) {

   if(argc!=4) {
      printf("Invalid Syntax <Nodecount> <Edgecount> <Itercount>\n");
      return 0;
   }
   int i = 0, j = 0, k = 0, run = 0, idx = 0;
   int nodes=atoi(argv[1]);
   int edges=atoi(argv[2]);
   int iter = atoi(argv[3]);
   struct timeval stop, start;

   int* indegree_count=allocateMemoryInt(nodes);
   int* outdegree_count=allocateMemoryInt(nodes);
   int* running_edge_indices=allocateMemoryInt(nodes);
   int* edges_1D = allocateMemoryInt(edges);//node1:node2|node2->node1

   double* pagerank_new;
   double* pagerank_old;

   if ((pagerank_new = (double *)malloc(nodes * sizeof(double))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }

   if ((pagerank_old = (double *)malloc(nodes * sizeof(double))) == NULL) {
      fprintf(stderr, "MALLOC ERROR: %s\n", strerror(errno));
      exit(1);
   }
   
   memset(pagerank_new, 0, nodes * sizeof(double));
   
   // set starting values for pagerank values to 1/n
   for (i = 0; i < nodes; i++)
      pagerank_old[i] = 1.0 / (double)nodes;
   
   gettimeofday(&start, NULL);
   setvbuf(stdin, NULL, _IOFBF, edges);

   //reads in edges (the nodes that point to other nodes)
   for (i = 0; i < edges; i++) {
      scanf("%d\n", &j);
      edges_1D[i] = j;
   }

   //reads in in-degrees, out-degrees, and computes value for running idx array
   for (i = 0; i < nodes; i++) {
      scanf("%d %d %d\n", &idx, &j, &k);
      indegree_count[idx] = j;
      outdegree_count[idx] = k;
      running_edge_indices[idx] = run;
      run += j;
   }

   gettimeofday(&stop, NULL);
   fprintf(stderr, "Read took %lf seconds\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));

   // Begin Cuda Setup 
   int *in_d, *out_d, *run_d, *edges_d;
   double *pagerank_new_d, *pagerank_old_d;
   
   int node_size = nodes * sizeof(int);
   int pr_size = nodes * sizeof(double);
   int edges_size = edges * sizeof(int);
  
   hipMalloc(&in_d, node_size);
   hipMemcpy(in_d, indegree_count, node_size, hipMemcpyHostToDevice);
   
   hipMalloc(&out_d, node_size);
   hipMemcpy(out_d, outdegree_count, node_size, hipMemcpyHostToDevice);
   
   hipMalloc(&run_d, node_size);
   hipMemcpy(run_d, running_edge_indices, node_size, hipMemcpyHostToDevice);
   
   hipMalloc(&edges_d, edges_size);
   hipMemcpy(edges_d, edges_1D, edges_size, hipMemcpyHostToDevice);
   
   hipMalloc(&pagerank_old_d,pr_size);
   hipMemcpy(pagerank_old_d, pagerank_old, pr_size, hipMemcpyHostToDevice);
   
   hipMalloc(&pagerank_new_d,pr_size);

   // Computes number of blocks needed, creates a "row-like" grid with "row-like" blocks
   int blocks = ceil((double)nodes/(double)BLOCKWIDTH);
   dim3 dimGrid(blocks, 1, 1);
   dim3 dimBlock(BLOCKWIDTH, 1, 1);

   gettimeofday(&start, NULL);

   for(i=0; i < iter; i++) {
	   CalcPageRank<<<dimGrid, dimBlock>>>(nodes, edges, in_d, out_d, run_d, edges_d, pagerank_old_d, pagerank_new_d);	    
   }

   hipMemcpy(pagerank_old, pagerank_old_d, nodes * sizeof(double), hipMemcpyDeviceToHost);

   gettimeofday(&stop, NULL);
   fprintf(stderr, "Compute took %lf seconds\n\n", (stop.tv_sec - start.tv_sec) +
      ((stop.tv_usec - start.tv_usec) / 1000000.0));

   hipFree(in_d);
   hipFree(out_d);
   hipFree(run_d);
   hipFree(edges_d);
   hipFree(pagerank_old_d);
   hipFree(pagerank_new_d);

   // prints out all pagerank values and their respective nodes -> captured by python program
   for (i = 0; i < nodes; i++)
      printf("%.15lf:%d,", pagerank_old[i], i);

   free(indegree_count);
   free(outdegree_count);
   free(pagerank_new);
   free(pagerank_old);
   free(running_edge_indices);
   free(edges_1D);
   return 0;
}






